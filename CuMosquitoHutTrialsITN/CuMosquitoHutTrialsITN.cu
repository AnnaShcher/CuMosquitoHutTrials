#include "hip/hip_runtime.h"
#include "CuMosquitoHutTrialsITN.h"
#pragma comment(lib, "cudart")

#include <ctime>


#define PI 3.14159265359
#define e 2.718281828459
#define FINAL_TIME 36000//duration of experiment in seconds
#define NITERATIONS 18000
#define NSTEPS 19000
#define HUT_SIZE 1.5	//meters
#define NET_SIZE 0.75
#define STD_A 80/3.0	//std of attraction kernel
#define EPS 2.2204e-16
#define CONC_NET_SURF pow(e,(-0.5*pow(NET_SIZE,2.0)/pow(STD_A,2.0)))	//concentration of CO2 on net surface
#define CUDA_BLOCK_DIM	16	//dimension of a single rectangular CUDA execution block
#define CUDA_BLOCK_SIZE CUDA_BLOCK_DIM*CUDA_BLOCK_DIM	//number of CUDA threads in a single block

template<typename T>
T min(T a, T b){ return a<b ? a : b; }
template<typename T>
T max(T a, T b){ return a>b ? a : b; }


__global__ void setup_kernel(RandomGeneratorState* p_rnd_states, uint64_t seed_spin_up, uint32_t repetitions, uint32_t experiments)
{
	unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (id_x >= experiments || id_y >= repetitions) return;

	unsigned int id = id_y * experiments + id_x;

	/* Each thread gets same seed, a different sequence number,
	no offset */
	hiprand_init(seed_spin_up, id + 0, 0, &p_rnd_states[id].state_xinit);
	hiprand_init(seed_spin_up, id + 1, 0, &p_rnd_states[id].state_yinit);
	hiprand_init(seed_spin_up, id + 2, 0, &p_rnd_states[id].state_active);
	hiprand_init(seed_spin_up, id + 3, 0, &p_rnd_states[id].state_dead);
	hiprand_init(seed_spin_up, id + 4, 0, &p_rnd_states[id].state_d1);
	hiprand_init(seed_spin_up, id + 5, 0, &p_rnd_states[id].state_d2);
	hiprand_init(seed_spin_up, id + 6, 0, &p_rnd_states[id].state_th1);
	hiprand_init(seed_spin_up, id + 7, 0, &p_rnd_states[id].state_th2);
	hiprand_init(seed_spin_up, id + 8, 0, &p_rnd_states[id].state_attr);
	hiprand_init(seed_spin_up, id + 9, 0, &p_rnd_states[id].state_acc);
}


__global__ void hut_exp_kernel(char* in, char* dead, char* trap, char* fed, char* unf_dead, double* Ctot, double* net_cont,
	const hut_exp_ITN xin,
	RandomGeneratorState* p_rnd_states,
	uint32_t repetitions, uint32_t experiments)
{
	unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (id_x >= experiments || id_y >= repetitions) return;

	unsigned int id = id_y * experiments + id_x;
    unsigned int id_CTOT, id_ncont;

	bool entered, out;
	int tmax = (int)(3600 * xin.tmax);
	int tindrs = 0;
    int step = 0;
	bool  attr, acc, move;
	double death, p_attr, p_rep;
	double death_rate = xin.mu / 1800 / 34;
	double d1, th1, d2, th2;
	double x, y, xnew, ynew, dold, dnew, cold, cnew;
	double sig_acc, conc_chem = 0.0, count_ncont = 0.0, death_rate_psn;
    double sig_max = xin.sig_acc[0] + xin.sig_acc[1] * 80.0;
	bool cond[7] = { false };

	x = xin.xlim[0] + (xin.xlim[1] - xin.xlim[0]) * hiprand_uniform(&p_rnd_states[id].state_xinit);
	y = xin.ylim[0] + (xin.ylim[1] - xin.ylim[0]) * hiprand_uniform(&p_rnd_states[id].state_yinit);
	dold = sqrt(pow(x, 2.0) + pow(y, 2.0));
	while (dold < NET_SIZE)
	{
		x = xin.xlim[0] + (xin.xlim[1] - xin.xlim[0]) * hiprand_uniform(&p_rnd_states[id].state_xinit);
		y = xin.ylim[0] + (xin.ylim[1] - xin.ylim[0]) * hiprand_uniform(&p_rnd_states[id].state_yinit);
		dold = sqrt(pow(x, 2.0) + pow(y, 2.0));
	}
	cold = pow(e, (-0.5 * pow(dold, 2.0) / pow(STD_A, 2.0)));
	entered = dold < HUT_SIZE;
	cond[0] = entered;
	//variables, conditions
	//[0      1       2    3   4                5          
	//[In/Out Trapped Dead Fed taxis/kinesis  inside_net  


	for (int n = 2; n <= FINAL_TIME; n += 2)
	{
		//natural death
		death = hiprand_uniform_double(&p_rnd_states[id].state_dead);
		//death_rate_psn = (1 - death_rate) * pow(e, -1.0 / xin.alpha_p / conc_chem);
		//cond[2] = death < death_rate + death_rate_psn;//select the 'fortune'
        if (conc_chem == 0)
            cond[2] = death < death_rate;
        else
            //cond[2] = death < death_rate  + 1.0/(1.0 + pow(e,-(conc_chem - xin.alpha_p) / xin.alpha_d));//select the 'fortune'
            cond[2] = death < death_rate + xin.alpha_p*conc_chem;//select the 'fortune'
        //cond[2] = death < death_rate + 1.0 / (1.0 + exp(-(conc_chem - xin.alpha_d) / xin.alpha_p));
		//if (cond[2])
		//	break;

		move = !cond[1] && !cond[2];	//not trapped,& not dead, & not resting

		//candidate step
		d1 = 0.4 + 0.1* hiprand_normal_double(&p_rnd_states[id].state_d1);
		th1 = 2 * PI * hiprand_uniform(&p_rnd_states[id].state_th1);
		d2 = 0.4 + 0.1* hiprand_normal_double(&p_rnd_states[id].state_d1);
		th2 = 2 * PI * hiprand_uniform(&p_rnd_states[id].state_th1);
		xnew = x + d1 * cos(th1) + d2 * cos(th2);
		ynew = y + d1 * sin(th1) + d2 * sin(th2);

		//measuring concentration for new position
		dnew = sqrt(pow(xnew, 2.0) + pow(ynew, 2.0));
		cnew = pow(e, (-0.5 * pow(dnew, 2.0) / pow(STD_A, 2.0)));
       
        sig_acc = xin.sig_acc[0] + xin.sig_acc[1] * dold;//scaling factor for attraction
        //sig_acc = xin.sig_acc[0] + xin.sig_acc[1] * dold;

		p_attr = min(1.0, pow(e, (cnew - cold) / sig_acc));//prob of acceptance
		if (!cond[5])
			p_rep = xin.r * min(1.0, (1.0 - 1.0 / (1.0 + pow(e, -(dnew - xin.d50) / xin.s))));//concentration of repellent

		else
			p_rep = xin.r * min(1.0, (1.0 / (1.0 + pow(e, -(dnew - xin.d50) / xin.s))));//concentration of repellent
		attr = cond[4] && hiprand_uniform_double(&p_rnd_states[id].state_attr) < (1 - p_rep) ||
			hiprand_uniform_double(&p_rnd_states[id].state_attr) < p_attr * (1 - p_rep);//attraction or kinesis

        id_CTOT = repetitions * experiments* step + id_y * experiments + id_x;
        id_ncont = id * NITERATIONS + step;
        //Ctot[id_CTOT] = conc_chem;
        //net_cont[id_CTOT] = count_ncont;
        Ctot[id_CTOT] = xin.d50_NetCont;
        net_cont[id_CTOT] = xin.s_NetCont;
        step = step + 1;
        conc_chem = conc_chem -xin.d50_NetCont*conc_chem;

		if (!cond[5] && dnew <= NET_SIZE)
		{
			acc = hiprand_uniform_double(&p_rnd_states[id].state_acc) < 1.0 - xin.pnet;
			if (move && attr && !acc)
			{
				x = NET_SIZE + EPS;
				y = 0;//hiting the sufice
				cold = CONC_NET_SURF;
				dold = NET_SIZE + EPS;
				conc_chem = conc_chem + xin.r * min(1.0, (1.0 - 1.0 / (1.0 + pow(e, -(dold - xin.d50) / xin.s))));
                count_ncont = count_ncont + 1.0;
			}
		}
		else if (cond[0] && dnew > HUT_SIZE)
			acc = hiprand_uniform_double(&p_rnd_states[id].state_active) < xin.phut;
		else
			acc = true;//taking into account net barrier
		if (move && attr && acc)
		{
			x = xnew;
			y = ynew;
			cold = cnew; //and resp. site & conc. values
			dold = dnew;
		}
		out = !cond[0];
		cond[0] = cond[0] || (dold <= HUT_SIZE);//inside
		entered = out && cond[0];
		cond[1] = cond[1] || (cond[0] && (dold >= HUT_SIZE));//mark trapped mosquitoes
		cond[3] = cond[3] || (dold < xin.eps);
		cond[5] = dold < NET_SIZE;
		if (cond[0] && !cond[2] && !cond[1])
			tindrs = tindrs + 2;//increment time spent indoors
		cond[4] = cond[4] || (tindrs >= tmax) || cond[3];
	}
	death = hiprand_uniform_double(&p_rnd_states[id].state_dead);
	//cond[2] = cond[2] || (death < 24 * xin.mu / 34 + xin.alpha_p * pow(conc,0.7) * 24 * 1800);	//select the 'fortune'
	//death_rate_psn = (1 - death_rate) * pow(e, -1.0 / xin.alpha_p / conc_chem);
    //cond[2] = cond[2] || (death < 24 * xin.mu / 34 + death_rate_psn * 24 * 1800);	//select the 'fortune'
    if(conc_chem == 0)
        cond[2] = cond[2] || (death < 24 * xin.mu / 34);	//select the 'fortune'
    else
	    //cond[2] = cond[2] || (death < 24 * xin.mu / 34 + 1.0 / (1.0 + pow(e,-(conc_chem - xin.alpha_p) / xin.alpha_d)) * 24.0 * 1800.0);	//select the 'fortune
        cond[2] = cond[2] || (death < 24 * xin.mu / 34 + +xin.alpha_p*conc_chem * 24.0 * 1800.0);	//select the 'fortune'
	in[id] = cond[0];
	dead[id] = cond[0] && cond[2];
	trap[id] = cond[1];
	fed[id] = cond[3];
	unf_dead[id] = cond[2] && !cond[3];
	/* Copy state back to global memory */
	/* Store results */
}


inline void setupGrid(uint32_t thread_row_num, uint32_t thread_col_num, dim3& grid_dim, dim3& block_dim)
{
	block_dim = dim3(CUDA_BLOCK_DIM, CUDA_BLOCK_DIM);

	uint32_t grid_dim_y = (thread_row_num % CUDA_BLOCK_DIM == 0) ?
		thread_row_num / CUDA_BLOCK_DIM :
		(thread_row_num & ~(CUDA_BLOCK_DIM - 1)) / CUDA_BLOCK_DIM + 1;

	uint32_t grid_dim_x = (thread_col_num % CUDA_BLOCK_DIM == 0) ?
		thread_col_num / CUDA_BLOCK_DIM :
		(thread_col_num & ~(CUDA_BLOCK_DIM - 1)) / CUDA_BLOCK_DIM + 1;
	grid_dim = dim3(grid_dim_x, grid_dim_y);
}


void initialize(uint32_t repetitions, uint32_t num_experiments, ComputationContext& cmpt_ctx)
{
	dim3 grid_dim, block_dim;
	setupGrid(repetitions, num_experiments, grid_dim, block_dim);

	cmpt_ctx.hostIn = (char*)malloc(repetitions * num_experiments);
	cmpt_ctx.hostDead = (char*)malloc(repetitions * num_experiments);
	cmpt_ctx.hostTrap = (char*)malloc(repetitions * num_experiments);
	cmpt_ctx.hostFed = (char*)malloc(repetitions * num_experiments);
	cmpt_ctx.hostUnfDead = (char*)malloc(repetitions * num_experiments);
	cmpt_ctx.hostCtot = (double*)malloc(repetitions * num_experiments * NITERATIONS*sizeof(double));
    cmpt_ctx.hostNetCont = (double*)malloc(repetitions * num_experiments * NITERATIONS * sizeof(double));
	/* Set results to 0 */
	/* Allocate space for prng states on device */

	hipSetDevice(0);
	hipDeviceReset();

	hipMalloc(&cmpt_ctx.p_rnd_states, repetitions * num_experiments * sizeof(RandomGeneratorState));

	/* Setup prng states */
	setup_kernel << <grid_dim, block_dim >> >(cmpt_ctx.p_rnd_states, time(NULL), repetitions, num_experiments);

	hipMalloc(&cmpt_ctx.devIn, repetitions * num_experiments);
	hipMalloc(&cmpt_ctx.devDead, repetitions * num_experiments);
	hipMalloc(&cmpt_ctx.devTrap, repetitions * num_experiments);
	hipMalloc(&cmpt_ctx.devFed, repetitions * num_experiments);
	hipMalloc(&cmpt_ctx.devUnfDead, repetitions * num_experiments);
	hipMalloc(&cmpt_ctx.devCtot, repetitions * num_experiments* NITERATIONS * sizeof(double));
    hipMalloc(&cmpt_ctx.devNetCont, repetitions * num_experiments * NITERATIONS * sizeof(double));
}


void CallKernel(uint32_t repetitions, uint32_t num_experiments, const hut_exp_ITN& experimental_params, const ComputationContext& cmpt_ctx)
{
	if (hipGetLastError() != hipSuccess) return;

	dim3 grid_dim, block_dim;
	setupGrid(repetitions, num_experiments, grid_dim, block_dim);

	hut_exp_kernel << <grid_dim, block_dim >> >(cmpt_ctx.devIn, cmpt_ctx.devDead, cmpt_ctx.devTrap, cmpt_ctx.devFed, cmpt_ctx.devUnfDead, cmpt_ctx.devCtot, cmpt_ctx.devNetCont,
		experimental_params, cmpt_ctx.p_rnd_states, repetitions, num_experiments);

	hipMemcpy(cmpt_ctx.hostIn, cmpt_ctx.devIn, repetitions*num_experiments, hipMemcpyDeviceToHost);
	hipMemcpy(cmpt_ctx.hostDead, cmpt_ctx.devDead, repetitions*num_experiments, hipMemcpyDeviceToHost);
	hipMemcpy(cmpt_ctx.hostTrap, cmpt_ctx.devTrap, repetitions*num_experiments, hipMemcpyDeviceToHost);
	hipMemcpy(cmpt_ctx.hostFed, cmpt_ctx.devFed, repetitions*num_experiments, hipMemcpyDeviceToHost);
	hipMemcpy(cmpt_ctx.hostUnfDead, cmpt_ctx.devUnfDead, repetitions*num_experiments, hipMemcpyDeviceToHost);
	hipMemcpy(cmpt_ctx.hostCtot, cmpt_ctx.devCtot, repetitions*num_experiments* NITERATIONS * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(cmpt_ctx.hostNetCont, cmpt_ctx.devNetCont, repetitions * num_experiments * NITERATIONS * sizeof(double), hipMemcpyDeviceToHost);
}


void shutdown(const ComputationContext& cmpt_ctx)
{
	//Free host memory
	free(cmpt_ctx.hostIn);
	free(cmpt_ctx.hostDead);
	free(cmpt_ctx.hostTrap);
	free(cmpt_ctx.hostFed);
	free(cmpt_ctx.hostUnfDead);
	free(cmpt_ctx.hostCtot);
    free(cmpt_ctx.hostNetCont);

	//Free GPU memory
	hipFree(cmpt_ctx.devIn);
	hipFree(cmpt_ctx.devDead);
	hipFree(cmpt_ctx.devTrap);
	hipFree(cmpt_ctx.devFed);
	hipFree(cmpt_ctx.devUnfDead);
	hipFree(cmpt_ctx.devCtot);
    hipFree(cmpt_ctx.devNetCont);
	hipFree(cmpt_ctx.p_rnd_states);

}


